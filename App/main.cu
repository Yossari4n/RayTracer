#include <RayTracer/host/Camera.h>
#include <RayTracer/host/BruteForce.h>
#include <RayTracer/host/BVH.h>
#include <RayTracer/host/KDTree.h>
#include <RayTracer/host/PPMTarget.h>
#include <RayTracer/host/Scene.h>

#include <RayTracer/device/Camera.cuh>
#include <RayTracer/device/PPMTarget.cuh>
#include <RayTracer/device/BruteForce.cuh>
#include <RayTracer/device/BVH.cuh>
#include <RayTracer/device/Scene.cuh>

#define TINYOBJLOADER_IMPLEMENTATION 
#include "../RayTracer/tiny_obj_loader.h"

#include <nlohmann/json.hpp>

#include <iostream>
#include <string>

namespace glm {

void from_json(const nlohmann::json& j, glm::vec3& v) {
    j.at("x").get_to(v.x);
    j.at("y").get_to(v.y);
    j.at("z").get_to(v.z);
}

}

struct RayGeneratorConfig {
    std::string name;
    glm::vec3 position;
    glm::vec3 lookAt;
    float fov;
};

struct RenderTargetConfig {
    std::string name;
    unsigned int width;
    unsigned int height;
};

struct AccelerationStructureConfig {
    std::string name;
};

struct Config {
    std::string scene;
    unsigned int samplesPerPixel;
    unsigned int maxDepth;
    bool cuda;
    RayGeneratorConfig rayGenerator;
    AccelerationStructureConfig accelerationStructure;
    RenderTargetConfig renderTarget;
};

void from_json(const nlohmann::json& json, Config& config) {
    json.at("scene").get_to(config.scene);
    json.at("samples_per_pixel").get_to(config.samplesPerPixel);
    json.at("max_depth").get_to(config.maxDepth);
    json.at("cuda").get_to(config.cuda);

    const auto& rayGenerator = json.at("ray_generator");
    rayGenerator.at("name").get_to(config.rayGenerator.name);
    rayGenerator.at("position").get_to(config.rayGenerator.position);
    rayGenerator.at("look_at").get_to(config.rayGenerator.lookAt);
    rayGenerator.at("fov").get_to(config.rayGenerator.fov);

    const auto& accelerationStructure = json.at("acceleration_structure");
    accelerationStructure.at("name").get_to(config.accelerationStructure.name);

    const auto& renderTarget = json.at("render_target");
    renderTarget.at("name").get_to(config.renderTarget.name);
    renderTarget.at("width").get_to(config.renderTarget.width);
    renderTarget.at("width").get_to(config.renderTarget.height);
}

void HostMain(const Config& config) {
    std::unique_ptr<rt::IRayGenerator> rayGenerator;
    if(config.rayGenerator.name == "Camera") {
        rayGenerator = std::make_unique<rt::Camera>(
            config.rayGenerator.position,
            config.rayGenerator.lookAt,
            rt::Vector3(0.0f, 1.0f, 0.0f),      // up
            config.rayGenerator.fov,
            16.0f / 9.0f,                       // aspect ratio
            0.1f,                               // aperture
            10.0f                               // focus_distance
        );
    }

    std::unique_ptr<rt::IAccelerationStructure> accelerationStructure;
    if(config.accelerationStructure.name == "BruteForce") {
        accelerationStructure = std::make_unique<rt::BruteForce>();
    } else if(config.accelerationStructure.name == "BVH") {
        accelerationStructure = std::make_unique<rt::BVH>();
    } else if(config.accelerationStructure.name == "KDTree") {
        accelerationStructure = std::make_unique<rt::KDTree>();
    }

    std::unique_ptr<rt::IRenderTarget> renderTarget;
    if(config.renderTarget.name == "PPMTarget") {
        renderTarget = std::make_unique<rt::PPMTarget>(
            config.renderTarget.width,
            config.renderTarget.height
        );
    }

    rt::Scene scene(
        rayGenerator.get(), 
        accelerationStructure.get(), 
        renderTarget.get()
    );

    scene.LoadScene(config.scene);
    scene.GenerateFrame(config.samplesPerPixel, config.maxDepth);
}

void DeviceMain(const Config& config) {
    std::unique_ptr<rt::device::IRayGenerator> rayGenerator;
    if(config.rayGenerator.name == "Camera") {
        rayGenerator = std::make_unique<rt::device::Camera>(
            config.rayGenerator.position,
            config.rayGenerator.lookAt,
            rt::Vector3(0.0f, 1.0f, 0.0f),      // up
            config.rayGenerator.fov,
            16.0f / 9.0f,                       // aspect ratio
            0.1f,                               // aperture
            10.0f                               // focus_distance
            );
    }

    std::unique_ptr<rt::device::IAccelerationStructure> accelerationStructure;
    if(config.accelerationStructure.name == "BruteForce") {
        accelerationStructure = std::make_unique<rt::device::BruteForce>();
    } else if(config.accelerationStructure.name == "BVH") {
        accelerationStructure = std::make_unique<rt::device::BVH>();
    } else if(config.accelerationStructure.name == "KDTree") {

    }

    std::unique_ptr<rt::device::IRenderTarget> renderTarget;
    if(config.renderTarget.name == "PPMTarget") {
        renderTarget = std::make_unique<rt::device::PPMTarget>(
            config.renderTarget.width,
            config.renderTarget.height
        );
    }

    rt::device::Scene scene(
        rayGenerator.get(),
        accelerationStructure.get(),
        renderTarget.get()
    );

    scene.LoadScene(config.scene);
    scene.GenerateFrame(config.samplesPerPixel, config.maxDepth, 8, 8);
}

void test(int* tree, int size, int predicate) {
    int i = 0;
    int leaf = 0;
    while(i < size) {
        printf("check: %d\n", tree[i]);
        int node = tree[i];
        if(tree[i] == predicate) {
            printf("found: %d\n", tree[i]);
        }

        if(i == size - 1) {
            return;
        }

        if(i < (size / 2) && tree[2 * i + 1] < 10) { // not leaf
            i = 2 * i + 1;
        } else {
            int k = 1;
            while(true) {
                i = (i - 1) / 2; // jump to the parent
                int p = k * 2;
                if(leaf % p == k - 1) break; // correct number of jumps found
                k = p;
            }
            // after we jumped to the parent, go to the right child
            i = 2 * i + 2;
            leaf++; // next leaf, please

            if(tree[2 * i + 1] >= 10 && tree[2 * i + 2] >= 10) {
                return;
            }
        }
    }
}

int main(int argc, char* argv[]) {

    //int* arr = new int[]{ 1, 2, 3, 4, 5, 16, 17 };
    //int* d_arr;
    //hipMalloc((void**)&d_arr, sizeof(int) * 7);
    //hipMemcpy(d_arr, arr, sizeof(int) * 7, hipMemcpyHostToDevice);
    ////test<<<1, 1>>>(d_arr, 7);
    //test(arr, 7, 6);
    //return;

    std::ifstream jsonFile(argv[1]);
    nlohmann::json configJson;
    jsonFile >> configJson;
    Config config = configJson.get<Config>();

    if(config.cuda) {
        DeviceMain(config);
    } else {
        HostMain(config);
    }

    return 0;
}

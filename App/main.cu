#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include <hiprand/hiprand_kernel.h>

#include <RayTracer/test.h>

#include <iostream>

int main() {
    cuda_test<<<1,1>>>();
    test();
    return 0;
}